
#include <hip/hip_runtime.h>
/*

	Implement your CUDA kernel in this file

*/
#define TILE_DIM 32

__global__ void mirror_boundaries(double *E_prev, const int n, const int m)
{
  int row = blockIdx.y*blockDim.y + threadIdx.y + 1;
  int col = blockIdx.x*blockDim.x + threadIdx.x + 1;

  if (col == 1) {
    E_prev[row*(n+2)] = E_prev[row*(n+2) + 2];
    E_prev[row*(n+2) + n + 1] = E_prev[row*(n+2) + n - 1];
  }
  if (row == 1) {
    E_prev[col] = E_prev[2*(n+2) + col];
    E_prev[(m+1)*(n+2) + col] = E_prev[(m-1)*(n+2) + col];
  }
}

__global__ void simulate(double *E, double *E_prev, double *R, const double alpha,
			 const int n, const int m, const double kk,
			 const double dt, const double a, const double epsilon,
			 const double M1,const double M2, const double b)
{
  // extern __shared__ double E_Block[];
  // extern __shared__ double R_Block[];

  // __shared__ double *E_Block;
  // __shared__ double *R_Block;

  // extern __shared__ double E_Block[];
  // extern __shared__ double R_Block[];

  __shared__ double E_Block[TILE_DIM*TILE_DIM];
  __shared__ double R_Block[TILE_DIM*TILE_DIM];

  int ty = threadIdx.y;
  int tx = threadIdx.x;

  int row = blockIdx.y*blockDim.y + ty + 1;
  int col = blockIdx.x*blockDim.x + tx + 1;

  E_Block[ty*TILE_DIM + tx] = E[row*(n+2) + col];
  R_Block[ty*TILE_DIM + tx] = R[row*(n+2) + col];

  if ((row - 1 < m) && (col - 1 < n)) {

    // E_Block[ty*TILE_DIM + tx] = E[row*(n+2) + col];
    // R_Block[ty*TILE_DIM + tx] = R[row*(n+2) + col];

    __syncthreads();

    E[row*(n+2)+col] = E_prev[row*(n+2)+col] + alpha*(E_prev[row*(n+2)+col+1] + E_prev[row*(n+2)+col-1] - 4*E_prev[row*(n+2)+col] + E_prev[(row+1)*(n+2)+col] + E_prev[(row-1)*(n+2)+col]);

    E[row*(n+2) + col] = E_Block[ty*TILE_DIM+tx] = E_Block[ty*TILE_DIM+tx] - dt*(kk*E_Block[ty*TILE_DIM+tx]*(E_Block[ty*TILE_DIM+tx] - a)*(E_Block[ty*TILE_DIM+tx] - 1) + E_Block[ty*TILE_DIM+tx]*R_Block[ty*TILE_DIM+tx]);
    R[row*(n+2) + col] = R_Block[ty*TILE_DIM+tx] + dt*(epsilon + M1*R_Block[ty*TILE_DIM+tx]/(E_Block[ty*TILE_DIM+tx] + M2))*(-R_Block[ty*TILE_DIM+tx] - kk*E_Block[ty*TILE_DIM+tx]*(E_Block[ty*TILE_DIM+tx] - b - 1));

  }
}
